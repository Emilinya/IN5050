#include <hip/hip_runtime.h>

#include "utils.h"
#include "tables.h"

// Read planar YUV frames with 4:2:0 chroma sub-sampling */
__host__ yuv_t *read_yuv(FILE *file, struct c63_common *cm)
{
    size_t len = 0;
    yuv_t *image;
    cudaMallocErr(image, sizeof(yuv_t));

    /* Read Y. The size of Y is the same as the size of the image. The indices
       represents the color component (0 is Y, 1 is U, and 2 is V) */
    cudaCallocErr(image->Y, 1, cm->ypw * cm->yph);
    len += fread(image->Y, 1, cm->width * cm->height, file);

    /* Read U. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y
       because (height/2)*(width/2) = (height*width)/4. */
    cudaCallocErr(image->U, 1, cm->upw * cm->uph);
    len += fread(image->U, 1, (cm->width * cm->height) / 4, file);

    /* Read V. Given 4:2:0 chroma sub-sampling, the size is 1/4 of Y. */
    cudaCallocErr(image->V, 1, cm->vpw * cm->vph);
    len += fread(image->V, 1, (cm->width * cm->height) / 4, file);

    if (ferror(file))
    {
        perror("ferror");
        exit(EXIT_FAILURE);
    }

    if (feof(file))
    {
        free_yuv(image);
        return NULL;
    }
    else if (len != cm->width * cm->height * 1.5)
    {
        fprintf(stderr, "Reached end of file, but incorrect bytes read.\n");
        fprintf(stderr, "Wrong input? (height: %d width: %d)\n", cm->height, cm->width);

        free_yuv(image);
        return NULL;
    }

    return image;
}

__host__ yuv_t *create_yuv(struct c63_common *cm)
{
    yuv_t *yuv;
    
    cudaMallocErr(yuv, sizeof(yuv_t));
    cudaCallocErr(yuv->Y, cm->ypw * cm->yph, sizeof(uint8_t));
    cudaCallocErr(yuv->U, cm->upw * cm->uph, sizeof(uint8_t));
    cudaCallocErr(yuv->V, cm->vpw * cm->vph, sizeof(uint8_t));

    return yuv;
}

__host__ void free_yuv(yuv_t *image)
{
    hipFree(image->Y);
    hipFree(image->U);
    hipFree(image->V);
    hipFree(image);
}

__host__ struct frame *create_frame(struct c63_common *cm)
{
    struct frame *f;
    cudaMallocErr(f, sizeof(struct frame));
    
    f->recons = create_yuv(cm);
    f->predicted = create_yuv(cm);

    cudaMallocErr(f->residuals, sizeof(dct_t));
    cudaCallocErr(f->residuals->Ydct, cm->ypw * cm->yph, sizeof(int16_t));
    cudaCallocErr(f->residuals->Udct, cm->upw * cm->uph, sizeof(int16_t));
    cudaCallocErr(f->residuals->Vdct, cm->vpw * cm->vph, sizeof(int16_t));

    cudaCallocErr(
        f->mbs[Y_COMPONENT],
        cm->mb_rows * cm->mb_cols, sizeof(struct macroblock));
    cudaCallocErr(
        f->mbs[U_COMPONENT],
        cm->mb_rows / 2 * cm->mb_cols / 2, sizeof(struct macroblock));
    cudaCallocErr(
        f->mbs[V_COMPONENT],
        cm->mb_rows / 2 * cm->mb_cols / 2, sizeof(struct macroblock));

    return f;
}

__host__ void free_frame(struct frame *f)
{
    /* First frame doesn't have a reconstructed frame to destroy */
    if (!f)
    {
        return;
    }

    free_yuv(f->recons);
    free_yuv(f->predicted);

    hipFree(f->residuals->Ydct);
    hipFree(f->residuals->Udct);
    hipFree(f->residuals->Vdct);
    hipFree(f->residuals);

    hipFree(f->mbs[Y_COMPONENT]);
    hipFree(f->mbs[U_COMPONENT]);
    hipFree(f->mbs[V_COMPONENT]);

    hipFree(f);
}

__host__ void init_tables(struct c63_common *cm) {
    // initialize quantization tables
    cudaMallocErr(cm->quanttbl[Y_COMPONENT], 64);
    cudaMallocErr(cm->quanttbl[U_COMPONENT], 64);
    cudaMallocErr(cm->quanttbl[V_COMPONENT], 64);
    for (int i = 0; i < 64; ++i)
    {
        cm->quanttbl[Y_COMPONENT][i] = yquanttbl_def[i] / (cm->qp / 10.0);
        cm->quanttbl[U_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
        cm->quanttbl[V_COMPONENT][i] = uvquanttbl_def[i] / (cm->qp / 10.0);
    }

    // initialize dct lookup
    cudaMallocErr(cm->dctlookup, 64 * sizeof(float));
    for (int i = 0; i < 64; ++i)
    {
        cm->dctlookup[i] = dct_lookup_table[i / 8][i % 8];
    }

    // initialize zigzag tables
    cudaMallocErr(cm->zigzag_U, 64);
    cudaMallocErr(cm->zigzag_V, 64);
    for (int i = 0; i < 64; ++i)
    {
        cm->zigzag_U[i] = zigzag_U_table[i];
        cm->zigzag_V[i] = zigzag_V_table[i];
    }
}

__host__ void free_tables(struct c63_common *cm) {
    hipFree(cm->quanttbl[Y_COMPONENT]);
    hipFree(cm->quanttbl[U_COMPONENT]);
    hipFree(cm->quanttbl[V_COMPONENT]);
    hipFree(cm->dctlookup);
    hipFree(cm->zigzag_U);
    hipFree(cm->zigzag_V);
}
