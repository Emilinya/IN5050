#include <hip/hip_runtime.h>

#include "c63.h"
#include "utils.h"
#include "gpu_utils.h"

__host__ struct gpu_frame *gpu_init(struct c63_common *cm)
{
    struct gpu_frame *frame;
    hipMallocManaged((void **)&frame, sizeof(struct gpu_frame));

    // allocate memory on the device for the input
    hipMallocManaged((void **)&frame->input, sizeof(yuv_t));
    hipMallocManaged((void **)&frame->input->Y, cm->yph * cm->ypw);
    hipMallocManaged((void **)&frame->input->U, cm->uph * cm->upw);
    hipMallocManaged((void **)&frame->input->V, cm->vph * cm->upw);
    
    // Allocate memory on the device for the reference
    hipMallocManaged((void **)&frame->reference, sizeof(yuv_t));
    hipMallocManaged((void **)&frame->reference->Y, cm->yph * cm->ypw);
    hipMallocManaged((void **)&frame->reference->U, cm->uph * cm->upw);
    hipMallocManaged((void **)&frame->reference->V, cm->vph * cm->upw);

    return frame;
}

__host__ void gpu_cleanup(struct gpu_frame *gpu_frame)
{
    hipFree(gpu_frame->input->Y);
    hipFree(gpu_frame->input->U);
    hipFree(gpu_frame->input->V);
    hipFree(gpu_frame->input);

    hipFree(gpu_frame->reference->Y);
    hipFree(gpu_frame->reference->U);
    hipFree(gpu_frame->reference->V);
    hipFree(gpu_frame->reference);

    hipFree(gpu_frame);
}
