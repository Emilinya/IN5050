#include "hip/hip_runtime.h"
#include <assert.h>
#include <errno.h>
#include <getopt.h>
#include <limits.h>
#include <math.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "cosine_transform.h"
#include "motion_estimate.h"
#include "utils.h"

/* Motion estimation for 8x8 block */
__global__ void me_block_8x8(
    int w, int h, uint8_t *orig, uint8_t *ref, struct macroblock *mbs)
{
    int mb_x = blockIdx.x;
    int mb_y = blockIdx.y;

    int range = blockDim.x / 2;

    // make sure we are within bounds of reference frame. TODO: Support partial frame bounds.
    int left = MAX(mb_x * 8 - range, 0);
    int top = MAX(mb_y * 8 - range, 0);
    int right = MIN(mb_x * 8 + range, w - 8);
    int bottom = MIN(mb_y * 8 + range, h - 8);

    int bounds_w = right - left;
    int bounds_h = bottom - top;
    int size = bounds_w * bounds_h;

    int tid = threadIdx.x + threadIdx.y * bounds_w;

    int mx = mb_x * 8;
    int my = mb_y * 8;

    // I would like not to hardcode this, but oh well 
    __shared__ uint16_t sad_grid[32 * 32];

    if (threadIdx.x < bounds_w && threadIdx.y < bounds_h) {
        uint8_t *origin = orig + mx + my * w;
        uint8_t *reference = ref + (left + threadIdx.x) + (top + threadIdx.y) * w;
    
        uint16_t abssum = 0;
        for (int v = 0; v < 8; ++v)
        {
            uint32_t a = 
                (origin[v * w] << 24) | (origin[v * w + 1] << 16)
                | (origin[v * w + 2] << 8)  | origin[v * w + 3];
            uint32_t b =
                (reference[v * w] << 24) | (reference[v * w + 1] << 16)
                | (reference[v * w + 2] << 8)  | reference[v * w + 3];
            uint32_t res = __vabsdiffu4(a, b);

            abssum += (res & (255u << 24)) >> 24;
            abssum += (res & (255u << 16)) >> 16;
            abssum += (res & (255u << 8)) >> 8;
            abssum += res & 255u;

            a = 
                (origin[v * w + 4] << 24) | (origin[v * w + 5] << 16)
                | (origin[v * w + 6] << 8)  | origin[v * w + 7];
            b =
                (reference[v * w + 4] << 24) | (reference[v * w + 5] << 16)
                | (reference[v * w + 6] << 8)  | reference[v * w + 7];
            res = __vabsdiffu4(a, b);

            abssum += (res & (255u << 24)) >> 24;
            abssum += (res & (255u << 16)) >> 16;
            abssum += (res & (255u << 8)) >> 8;
            abssum += res & 255u;
        }
        sad_grid[tid] = abssum;
    }
    
    __syncthreads();

    // optimal reduction number is sqrt(size) - set reduction number
    // to first order taylor expansion of sqrt(x) around 640
    const int reduction_num = 0.01976423537605 * (float)size + 12.64911064067;
    __shared__ uint16_t best_i_list[32];

    if (tid < reduction_num) {
        int start = tid * size / reduction_num;
        int end = (tid + 1) * size / reduction_num;

        int best_i = start;
        for (int i = start + 1; i < end; ++i)
        {
            if (sad_grid[i] < sad_grid[best_i])
            {
                best_i = i;
            }
        }
        best_i_list[tid] = best_i;
    }

    __syncthreads();

    if (tid == 0) {
        int best_i = best_i_list[0];
        for (int i = 1; i < reduction_num; ++i)
        {
            if (sad_grid[best_i_list[i]] < sad_grid[best_i])
            {
                best_i = best_i_list[i];
            }
        }

        struct macroblock *mb = &mbs[mb_x + mb_y * w / 8];
        mb->mv_x = left + (best_i % bounds_w) - mx;
        mb->mv_y = top + (best_i / bounds_w) - my;
        mb->use_mv = 1;
    }
}

__host__ void c63_motion_estimate(struct c63_common *cm)
{
    // define block grid
    dim3 block_grid_Y(cm->mb_cols, cm->mb_rows, 1);
    dim3 block_grid_UV(cm->mb_cols / 2, cm->mb_rows / 2, 1);
    
    // define thread grid
    dim3 thread_grid_Y(cm->me_search_range * 2, cm->me_search_range * 2, 1);
    dim3 thread_grid_UV(cm->me_search_range, cm->me_search_range, 1);

    // define streams
    hipStream_t Ystream, Ustream, Vstream;
    hipStreamCreate(&Ystream);
    hipStreamCreate(&Ustream);
    hipStreamCreate(&Vstream);

    // TODO: do something to properly use streams

    // Luma
    me_block_8x8 <<<block_grid_Y, thread_grid_Y, 0, Ystream>>> (
        cm->ypw, cm->yph,
        cm->curframe->orig->Y, cm->ref_recons->Y,
        cm->curframe->mbs[Y_COMPONENT]);
    hipStreamSynchronize(Ystream);
    // exit(1);
        
    // Chroma U
    me_block_8x8 <<<block_grid_UV, thread_grid_UV, 0, Ustream>>> (
        cm->upw, cm->uph,
        cm->curframe->orig->U, cm->ref_recons->U,
        cm->curframe->mbs[U_COMPONENT]);
    hipStreamSynchronize(Ustream);

    // Chroma V
    me_block_8x8 <<<block_grid_UV, thread_grid_UV, 0, Vstream>>> (
        cm->vpw, cm->vph,
        cm->curframe->orig->V, cm->ref_recons->V,
        cm->curframe->mbs[V_COMPONENT]);
    hipStreamSynchronize(Vstream);
}
